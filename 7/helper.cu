#include "hip/hip_runtime.h"
#include<iostream>
#include<conio.h>
#include<hip/hip_runtime.h>
#include<ctime>
#include<hip/hip_runtime.h>
#include<>
#include<hip/device_functions.h>
#include<iomanip>

#include "helper.cuh"

//using namespace std;//sil

__global__ void convolutionKernel(float* inputImage, float* filter, float* outputImage, int* filterSize, int inputWidth, int outputWidth, int outputHeight, int strideRow, int strideCol) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int fH = filterSize[0];
		int fW = filterSize[1];
		int outputIdx = r * outputWidth + c;
		int inputBeginX = c * strideCol;//kontrol et!!!
		int inputBeginY = r * strideRow;//kontrol et!!!

		int  fi = 0, fj = 0;
		float sum = 0;
		for (int i = inputBeginY; i < inputBeginY + fH; i++) {
			fj = 0;
			for (int j = inputBeginX; j < inputBeginX + fW; j++) {
				sum += inputImage[i * inputWidth + j] * filter[fi * fW + fj];
				fj++;
			}
			fi++;
		}
		outputImage[outputIdx] = sum;
	}
}

//float*
void convolutionCuda(float* inputImage, float* filter, int inputRow, int inputCol, int filterRow, int filterCol, int strideRow, int strideCol, float* outputImage, int outputRow, int outputCol) {
	int filterSize[] = { filterRow, filterCol };
	float* inputCuda, * filterCuda, * outputCuda;
	int* filterSizeCuda;

	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&filterCuda, sizeof(float) * filterRow * filterCol);
	hipMemcpy(filterCuda, filter, sizeof(float) * filterRow * filterCol, hipMemcpyHostToDevice);
	hipMalloc(&filterSizeCuda, sizeof(int) * 2);
	hipMemcpy(filterSizeCuda, filterSize, sizeof(int) * 2, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputRow * outputCol);

	dim3 gridDim(outputCol, outputRow, 1);
	convolutionKernel << <gridDim, 1 >> > (inputCuda, filterCuda, outputCuda, filterSizeCuda, inputCol, outputCol, outputRow, strideRow, strideCol);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputRow * outputCol, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(filterCuda);
	hipFree(filterSizeCuda);
	hipFree(outputCuda);

	//return outputImage;
}

__global__ void  convertToGrayKernel(float* inputImage, float* outputImage, int outputWidth, int outputHeight) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int inputRIdx = (r * outputWidth + c) * 3 + 0;
		int inputGIdx = (r * outputWidth + c) * 3 + 1;
		int inputBIdx = (r * outputWidth + c) * 3 + 2;

		outputImage[r * outputWidth + c] = (inputImage[inputRIdx] + inputImage[inputGIdx] + inputImage[inputBIdx]) / 3.0;
	}
}

//float*
void convertToGrayCuda(float* inputImage, int inputRow, int inputCol, float* outputImage) {
	int outputRow = inputRow, outputCol = inputCol;
	//float* outputImage = (float*)malloc(sizeof(float) * outputRow * outputCol);
	float* inputCuda, * outputCuda;

	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * 3);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * 3, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputRow * outputCol);

	dim3 gridDim(outputCol, outputRow, 1);
	convertToGrayKernel << <gridDim, 1 >> > (inputCuda, outputCuda, outputCol, outputRow);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputRow * outputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);

	//return outputImage;
}



__global__ void thresholdKernel(float* inputImage, float* outputImage, int thresh, int outputWidth, int outputHeight) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		if (inputImage[r * outputWidth + c] >= thresh) {
			outputImage[r * outputWidth + c] = 255;
		}
		else {
			outputImage[r * outputWidth + c] = 0;
		}
	}
}
void thresholdCuda(float* inputImage, float* outputImage, int thresh, int inputRow, int inputCol) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(inputCol, inputRow, 1);
	thresholdKernel << <gridDim, 1 >> > (inputCuda, outputCuda, thresh, inputCol, inputRow);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);

	//return outputImage;
}

__global__ void prewittAddKernel(float* inputImage1, float* inputImage2, float* outputImage, int outputHeight, int outputWidth) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		outputImage[r * outputWidth + c] = sqrt(inputImage1[r * outputWidth + c] * inputImage1[r * outputWidth + c] + inputImage2[r * outputWidth + c] * inputImage2[r * outputWidth + c]);
	}
}
void prewittAddCuda(float* inputImage1, float* inputImage2, float* outputImage, int inputRow, int inputCol) {
	float* inputCuda1, * inputCuda2, * outputCuda;
	hipMalloc(&inputCuda1, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda1, inputImage1, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&inputCuda2, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda2, inputImage2, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(inputCol, inputRow, 1);
	prewittAddKernel << <gridDim, 1 >> > (inputCuda1, inputCuda2, outputCuda, inputRow, inputCol);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda1);
	hipFree(inputCuda2);
	hipFree(outputCuda);
}
__global__ void absoluteValueKernel(float* inputImage, float* outputImage, int outputHeight, int outputWidth) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		if (inputImage[r * outputWidth + c] >= 0)
			outputImage[r * outputWidth + c] = inputImage[r * outputWidth + c];
		else
			outputImage[r * outputWidth + c] = -inputImage[r * outputWidth + c];
	}
}

void absoluteValueCuda(float* inputImage, float* outputImage, int inputRow, int inputCol) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(inputCol, inputRow, 1);
	absoluteValueKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputRow, inputCol);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}
__global__ void addArrayKernel(float* inputImage, float* sum, int outputHeight, int outputWidth) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		atomicAdd(sum, inputImage[r * outputWidth + c]);
		//sum[0] += inputImage[r * outputWidth + c];
	}
}

int addArrayCuda(float* inputImage, int outputHeight, int outputWidth) {
	float* outputImage = (float*)malloc(sizeof(float) * 1);
	outputImage[0] = 0;

	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * outputHeight * outputWidth);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * outputHeight * outputWidth, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * 1);
	hipMemcpy(outputCuda, outputImage, sizeof(float) * 1, hipMemcpyHostToDevice);

	dim3 gridDim(outputWidth, outputHeight, 1);
	addArrayKernel << <gridDim, 1 >> > (inputCuda, outputCuda, outputHeight, outputWidth);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * 1, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);

	return outputImage[0];
}

__global__ void scalePixelsKernel(float* inputImage, float* outputImage, int outputHeight, int outputWidth, float diff, float min, float newHigh, float newLow) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		outputImage[r * outputWidth + c] = ((inputImage[r * outputWidth + c] - min) / diff) * newHigh + newLow;
	}
}

void scalePixelsCuda(float* inputImage, float* outputImage, int outputHeight, int outputWidth, float diff, float min, float newHigh, float newLow) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * outputHeight * outputWidth);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * outputHeight * outputWidth, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputHeight * outputWidth);

	dim3 gridDim(outputWidth, outputHeight, 1);
	scalePixelsKernel << <gridDim, 1 >> > (inputCuda, outputCuda, outputHeight, outputWidth, diff, min, newHigh, newLow);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputHeight * outputWidth, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}

__global__ void resizeGrayImageKernel(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int outputHeight, int outputWidth) {
	//int(x_indis*eski_boyut_x/yeni_boyut_x),int(y_indis*eski_boyut_y / yeni_boyut_y)
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int x = (float)(c * inputWidth) / (float)outputWidth;
		int y = (float)(r * inputHeight) / (float)outputHeight;
		outputImage[r * outputWidth + c] = inputImage[y * inputWidth + x];
	}
}

__global__ void resizeRGBImageKernel(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int outputHeight, int outputWidth) {
	//int(x_indis*eski_boyut_x/yeni_boyut_x),int(y_indis*eski_boyut_y / yeni_boyut_y)
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int x = (float)(c * inputWidth) / (float)outputWidth;
		int y = (float)(r * inputHeight) / (float)outputHeight;
		outputImage[(r * outputWidth + c) * 3 + 0] = inputImage[(y * inputWidth + x) * 3 + 0];
		outputImage[(r * outputWidth + c) * 3 + 1] = inputImage[(y * inputWidth + x) * 3 + 1];
		outputImage[(r * outputWidth + c) * 3 + 2] = inputImage[(y * inputWidth + x) * 3 + 2];
	}
}

void resizeImageCuda(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int outputHeight, int outputWidth, int channel) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputHeight * inputWidth * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputHeight * inputWidth * channel, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputHeight * outputWidth * channel);

	dim3 gridDim(outputWidth, outputHeight, 1);
	if (channel == 1)
		resizeGrayImageKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputHeight, inputWidth, outputHeight, outputWidth);
	else if (channel == 3)
		resizeRGBImageKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputHeight, inputWidth, outputHeight, outputWidth);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputHeight * outputWidth * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}

__global__ void splitRGBChannelsKernel(float* inputImage, float* rImage, float* gImage, float* bImage, int inputHeight, int inputWidth) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputWidth && r < inputHeight) {
		rImage[r * inputWidth + c] = inputImage[(r * inputWidth + c) * 3 + 0];
		gImage[r * inputWidth + c] = inputImage[(r * inputWidth + c) * 3 + 1];
		bImage[r * inputWidth + c] = inputImage[(r * inputWidth + c) * 3 + 2];
	}
}

void splitRGBChannelsCuda(float* inputImage, float* rImage, float* gImage, float* bImage, int inputHeight, int inputWidth) {
	float* inputCuda, * rCuda, * gCuda, * bCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputHeight * inputWidth * 3);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputHeight * inputWidth * 3, hipMemcpyHostToDevice);
	hipMalloc(&rCuda, sizeof(float) * inputHeight * inputWidth);
	hipMalloc(&gCuda, sizeof(float) * inputHeight * inputWidth);
	hipMalloc(&bCuda, sizeof(float) * inputHeight * inputWidth);

	dim3 gridDim(inputWidth, inputHeight, 1);
	splitRGBChannelsKernel << <gridDim, 1 >> > (inputCuda, rCuda, gCuda, bCuda, inputHeight, inputWidth);

	hipMemcpy(rImage, rCuda, sizeof(float) * inputHeight * inputWidth, hipMemcpyDeviceToHost);
	hipMemcpy(gImage, gCuda, sizeof(float) * inputHeight * inputWidth, hipMemcpyDeviceToHost);
	hipMemcpy(bImage, bCuda, sizeof(float) * inputHeight * inputWidth, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(rCuda);
	hipFree(gCuda);
	hipFree(bCuda);
}

__global__ void addRGBChannelsKernel(float* inputImage, float* rImage, float* gImage, float* bImage, int inputHeight, int inputWidth) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputWidth && r < inputHeight) {
		inputImage[(r * inputWidth + c) * 3 + 0] = rImage[r * inputWidth + c];
		inputImage[(r * inputWidth + c) * 3 + 1] = gImage[r * inputWidth + c];
		inputImage[(r * inputWidth + c) * 3 + 2] = bImage[r * inputWidth + c];
	}
}

void addRGBChannelsCuda(float* inputImage, float* rImage, float* gImage, float* bImage, int inputHeight, int inputWidth) {
	float* inputCuda, * rCuda, * gCuda, * bCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputHeight * inputWidth * 3);
	hipMalloc(&rCuda, sizeof(float) * inputHeight * inputWidth);
	hipMemcpy(rCuda, rImage, sizeof(float) * inputHeight * inputWidth, hipMemcpyHostToDevice);
	hipMalloc(&gCuda, sizeof(float) * inputHeight * inputWidth);
	hipMemcpy(gCuda, gImage, sizeof(float) * inputHeight * inputWidth, hipMemcpyHostToDevice);
	hipMalloc(&bCuda, sizeof(float) * inputHeight * inputWidth);
	hipMemcpy(bCuda, bImage, sizeof(float) * inputHeight * inputWidth, hipMemcpyHostToDevice);

	dim3 gridDim(inputWidth, inputHeight, 1);
	addRGBChannelsKernel << <gridDim, 1 >> > (inputCuda, rCuda, gCuda, bCuda, inputHeight, inputWidth);

	hipMemcpy(inputImage, inputCuda, sizeof(float) * inputHeight * inputWidth * 3, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(rCuda);
	hipFree(gCuda);
	hipFree(bCuda);
}

__global__ void cutGrayImageKernel(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int beginY, int beginX, int endY, int endX) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputWidth && r < inputHeight && c < endX && c >= beginX && r < endY && r >= beginY) {
		int x = c - beginX, y = r - beginY, outputWidth = endX - beginX;
		outputImage[y * outputWidth + x] = inputImage[r * inputWidth + c];
	}
}

__global__ void cutRGBImageKernel(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int beginY, int beginX, int endY, int endX) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputWidth && r < inputHeight && c < endX && c >= beginX && r < endY && r >= beginY) {
		int x = c - beginX, y = r - beginY, outputWidth = endX - beginX;
		outputImage[(y * outputWidth + x) * 3 + 0] = inputImage[(r * inputWidth + c) * 3 + 0];
		outputImage[(y * outputWidth + x) * 3 + 1] = inputImage[(r * inputWidth + c) * 3 + 1];
		outputImage[(y * outputWidth + x) * 3 + 2] = inputImage[(r * inputWidth + c) * 3 + 2];
	}
}

void cutImageCuda(float* inputImage, float* outputImage, int inputHeight, int inputWidth, int beginY, int beginX, int endY, int endX, int channel) {
	int outputHeight = endY - beginY, outputWidth = endX - beginX;
	float* inputCuda, * outputCuda;
	dim3 gridDim(inputWidth, inputHeight, 1);
	hipMalloc(&inputCuda, sizeof(float) * inputHeight * inputWidth * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputHeight * inputWidth * channel, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputHeight * outputWidth * channel);
	if (channel == 1) {
		cutGrayImageKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputHeight, inputWidth, beginY, beginX, endY, endX);
	}
	else if (channel == 3) {
		cutRGBImageKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputHeight, inputWidth, beginY, beginX, endY, endX);
	}
	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputHeight * outputWidth * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}
__global__ void medianFilterMapKernel(float* inputImage, float* map, int fH, int fW, int inputWidth, int outputWidth, int outputHeight, int strideRow, int strideCol) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int outputIdx = r * outputWidth + c;

		int inputBeginX = c * strideCol;//kontrol et!!!
		int inputBeginY = r * strideRow;//kontrol et!!!				

		int  fi = 0, mWidth = fH * fW;
		for (int i = inputBeginY; i < inputBeginY + fH; i++) {
			for (int j = inputBeginX; j < inputBeginX + fW; j++) {
				map[outputIdx * mWidth + fi] = inputImage[i * inputWidth + j];
				fi++;
			}
		}
	}
}
__global__ void medianFilterSortKernel(float* inputImage, float* outputImage, float* map, int fH, int fW, int inputWidth, int outputWidth, int outputHeight, int strideRow, int strideCol) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < outputWidth && r < outputHeight) {
		int outputIdx = r * outputWidth + c;

		int inputBeginX = c * strideCol;//kontrol et!!!
		int inputBeginY = r * strideRow;//kontrol et!!!				

		int  mWidth = fH * fW;
		float temp;
		for (int i = 0; i < fH * fW - 1; i++) {
			for (int j = 0; j < fH * fW - i - 1; j++) {
				if (map[outputIdx * mWidth + j] > map[outputIdx * mWidth + j + 1]) {
					temp = map[outputIdx * mWidth + j + 1];
					map[outputIdx * mWidth + j + 1] = map[outputIdx * mWidth + j];
					map[outputIdx * mWidth + j] = temp;
				}
			}
		}

		int median = fW * fH / 2;
		outputImage[outputIdx] = map[outputIdx * mWidth + median];
	}
}

void medianFilterCuda(float* inputImage, int inputRow, int inputCol, int filterRow, int filterCol, int strideRow, int strideCol, float* outputImage, int outputRow, int outputCol) {
	float* inputCuda, * outputCuda, * mapCuda;

	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * outputRow * outputCol);
	hipMalloc(&mapCuda, sizeof(float) * inputRow * inputCol * filterRow * filterCol);

	dim3 gridDim(outputCol, outputRow, 1);
	medianFilterMapKernel << <gridDim, 1 >> > (inputCuda, mapCuda, filterRow, filterCol, inputCol, outputCol, outputRow, strideRow, strideCol);
	medianFilterSortKernel << <gridDim, 1 >> > (inputCuda, outputCuda, mapCuda, filterRow, filterCol, inputCol, outputCol, outputRow, strideRow, strideCol);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * outputRow * outputCol, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(outputCuda);
	hipFree(mapCuda);
}

__global__ void dilateKernel(float* inputImage, int inputRow, int inputCol, float* filter, int filterRow, int filterCol, float* outputImage) {
	int c = blockIdx.x, r = blockIdx.y;
	int idx = r * inputCol + c;
	if (c < inputCol && r < inputRow && inputImage[idx] == 255) {
		int fH2 = filterRow / 2, fW2 = filterCol / 2;
		if (r >= fH2 && c >= fW2 && r < inputRow - fH2 && c < inputCol - fW2) {
			int iterY = 0, iterX = 0;
			for (int i = r - fH2; i <= r + fH2; i++) {
				iterX = 0;
				for (int j = c - fW2; j <= c + fW2; j++) {
					if (filter[iterY * filterCol + iterX] == 255)
						outputImage[i * inputCol + j] = 255;
					iterX++;
				}
				iterY++;
			}
		}
	}
}

void dilateCuda(float* inputImage, int inputRow, int inputCol, float* filter, int filterRow, int filterCol, float* outputImage) {
	float* inputCuda, * outputCuda, * filterCuda;

	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&filterCuda, sizeof(float) * filterRow * filterCol);
	hipMemcpy(filterCuda, filter, sizeof(float) * filterRow * filterCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(outputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);

	dim3 gridDim(inputCol, inputRow, 1);
	dilateKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, filterCuda, filterRow, filterCol, outputCuda);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(filterCuda);
	hipFree(outputCuda);
}

__global__ void erodeKernel(float* inputImage, int inputRow, int inputCol, float* filter, int filterRow, int filterCol, float* outputImage) {
	int c = blockIdx.x, r = blockIdx.y;
	int idx = r * inputCol + c;
	if (c < inputCol && r < inputRow && inputImage[idx] == 255) {
		int fH2 = filterRow / 2, fW2 = filterCol / 2;
		if (r >= fH2 && c >= fW2 && r < inputRow - fH2 && c < inputCol - fW2) {
			bool same = true;
			int iterY = 0, iterX = 0;
			for (int i = r - fH2; i <= r + fH2; i++) {
				iterX = 0;
				for (int j = c - fW2; j <= c + fW2; j++) {
					if (filter[iterY * filterCol + iterX] == 255 && inputImage[i * inputCol + j] == 0) {
						same = false;
						break;
					}
					if (same != true) {
						break;
					}
					iterX++;
				}
				iterY++;
			}
			if (same == true)
				outputImage[idx] = 255;
		}
	}
}

void erodeCuda(float* inputImage, int inputRow, int inputCol, float* filter, int filterRow, int filterCol, float* outputImage) {
	float* inputCuda, * outputCuda, * filterCuda;

	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&filterCuda, sizeof(float) * filterRow * filterCol);
	hipMemcpy(filterCuda, filter, sizeof(float) * filterRow * filterCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);
	//hipMemcpy(outputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);

	dim3 gridDim(inputCol, inputRow, 1);
	erodeKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, filterCuda, filterRow, filterCol, outputCuda);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(filterCuda);
	hipFree(outputCuda);
}

__global__ void makeNumberGrayKernel(float* inputImage, int inputRow, int inputCol, float number) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int idx = r * inputCol + c;
		inputImage[idx] = number;
	}
}

__global__ void makeNumberRGBKernel(float* inputImage, int inputRow, int inputCol, float number1, float number2, float number3) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int idx = r * inputCol + c;
		inputImage[idx * 3 + 0] = number1;
		inputImage[idx * 3 + 1] = number2;
		inputImage[idx * 3 + 2] = number3;
	}
}

void makeNumberCuda(float* inputImage, int inputRow, int inputCol, int channel, float number1, float number2, float number3) {
	float* inputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);

	dim3 gridDim(inputCol, inputRow, 1);
	if (channel == 1)
		makeNumberGrayKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, number1);
	else if (channel == 3)
		makeNumberRGBKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, number1, number2, number3);

	hipMemcpy(inputImage, inputCuda, sizeof(float) * inputRow * inputCol * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
}

__global__ void histogramGrayKernel(float* inputImage, int inputRow, int inputCol, float* histogram) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int idx = r * inputCol + c;
		int hIdx = inputImage[idx];
		//histogram[hIdx] += 1;
		atomicAdd((histogram + hIdx), 1);
	}
}

__global__ void histogramRGBKernel(float* inputImage, int inputRow, int inputCol, float* histogram) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int idx = r * inputCol + c;
		int hIdx0 = inputImage[idx * 3 + 0];
		int hIdx1 = inputImage[idx * 3 + 1];
		int hIdx2 = inputImage[idx * 3 + 2];
		//histogram[hIdx0 * 3 + 0] += 1;
		//histogram[hIdx1 * 3 + 1] += 1;
		//histogram[hIdx2 * 3 + 2] += 1;
		atomicAdd((histogram + (hIdx2 * 3 + 0)), 1);
		atomicAdd((histogram + (hIdx2 * 3 + 1)), 1);
		atomicAdd((histogram + (hIdx2 * 3 + 2)), 1);
	}
}

void histogramCuda(float* inputImage, int inputRow, int inputCol, int channel, float* histogram) {
	float* inputCuda, * hisCuda;

	if (channel == 1) {
		for (int i = 0; i < 256; i++)
			histogram[i] = 0;
	}
	else {
		for (int i = 0; i < 256; i++) {
			histogram[i * 3 + 0] = 0;
			histogram[i * 3 + 1] = 0;
			histogram[i * 3 + 2] = 0;
		}
	}
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&hisCuda, sizeof(float) * 256 * channel);
	hipMemcpy(hisCuda, histogram, sizeof(float) * 256 * channel, hipMemcpyHostToDevice);

	dim3 gridDim(inputCol, inputRow, 1);
	if (channel == 1)
		histogramGrayKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, hisCuda);
	else if (channel == 3)
		histogramRGBKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, hisCuda);


	hipMemcpy(histogram, hisCuda, sizeof(float) * 256 * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(hisCuda);
}

__global__ void kmeansGrayClusterChoiceKernel(float* histogram, float* clusters, float* clusterCenters, float* errors, int k) {
	int c = blockIdx.x, r = blockIdx.y;
	if (histogram[r] > 0) {
		int min = (clusterCenters[0] - r) * (clusterCenters[0] - r), temp, minId = 0;
		for (int i = 1; i < k; i++) {
			temp = (clusterCenters[i] - r) * (clusterCenters[i] - r);
			if (temp < min) {
				min = temp;
				minId = i;
			}
		}
		clusters[r] = minId;
		//errors[minId] += min;
		errors[0] += min;
	}
}

__global__ void kmeansGrayCalcClusterCentersKernel(float* histogram, float* clusters, float* clusterCenters, float* clusterMeans, float* clusterElementCount, int k) {
	//int c = blockIdx.x, r = blockIdx.y;
	int c = threadIdx.x, r = threadIdx.y;
	if (histogram[r] > 0) {
		int id = clusters[r];
		atomicAdd((clusterMeans + id), r);
		atomicAdd((clusterElementCount + id), 1);
		//clusterMeans[id] += r;
		//clusterElementCount[id] += 1;
		__syncthreads();
		if (r < k) {
			if (clusterElementCount[r] != 0) {
				clusterMeans[r] /= clusterElementCount[r];
				clusterCenters[r] = clusterMeans[r];
			}
			clusterElementCount[r] = 0;
			clusterMeans[r] = 0;
		}
	}
}

__global__ void kmeansGrayOutputKernel(float* inputImage, int inputRow, int inputCol, float* outputImage, float* clusters, float* clusterCenters, int k) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int pixel = inputImage[r * inputCol + c];
		int cl = clusters[pixel];
		outputImage[r * inputCol + c] = clusterCenters[cl];
	}
}

__global__ void resetCECKernel(float* clusterElementCount, int k) {
	clusterElementCount[threadIdx.x] = 0;
}

void kmeansGrayCuda(float* inputImage, int inputRow, int inputCol, float* outputImage, int k) {
	//histogram calc
	float* histogram = (float*)malloc(sizeof(float) * 256);

	//cluster centers initialize
	float* clusters = (float*)malloc(sizeof(float) * 256);//256 color = cluster num
	float* clusterCenters = (float*)malloc(sizeof(float) * k);
	float* clusterElementCount = (float*)malloc(sizeof(float) * k);
	float* clusterMeans = (float*)malloc(sizeof(float) * k);
	float* clusterError = (float*)malloc(sizeof(float));//sum of all errors
	float errorOld = -1;
	clusterError[0] = 0;
	int mean = 256 / k;
	for (int i = 0; i < k; i++) {
		clusterCenters[i] = (i)*mean;
		clusterElementCount[i] = 0;
		clusterMeans[i] = 0;
	}
	for (int i = 0; i < 256; i++) {
		mean = rand() % k;
		histogram[i] = 0;
		clusters[i] = mean;
	}
	histogramCuda(inputImage, inputRow, inputCol, 1, histogram);

	float* inputCuda, * histCuda, * outputCuda, * cCuda, * cCCuda, * cECCuda, * cECuda, * cMCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&histCuda, sizeof(float) * 256);
	hipMemcpy(histCuda, histogram, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMalloc(&cCuda, sizeof(float) * 256);
	hipMemcpy(cCuda, clusters, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMalloc(&cCCuda, sizeof(float) * k);
	hipMemcpy(cCCuda, clusterCenters, sizeof(float) * k, hipMemcpyHostToDevice);
	hipMalloc(&cECCuda, sizeof(float) * k);
	hipMemcpy(cECCuda, clusterElementCount, sizeof(float) * k, hipMemcpyHostToDevice);
	hipMalloc(&cMCuda, sizeof(float) * k);
	hipMemcpy(cMCuda, clusterMeans, sizeof(float) * k, hipMemcpyHostToDevice);
	hipMalloc(&cECuda, sizeof(float));
	hipMemcpy(cECuda, clusterError, sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(inputCol, inputRow, 1);
	dim3 gridDim2(1, 256, 1);
	while (true) {
		kmeansGrayClusterChoiceKernel << <gridDim2, 1 >> > (histCuda, cCuda, cCCuda, cECuda, k);
		//kmeansGrayCalcClusterCentersKernel << <gridDim2, 1 >> > (histCuda, cCuda, cCCuda, cMCuda, cECCuda, k);
		kmeansGrayCalcClusterCentersKernel << <1, gridDim2 >> > (histCuda, cCuda, cCCuda, cMCuda, cECCuda, k);
		hipMemcpy(clusterError, cECuda, sizeof(float), hipMemcpyDeviceToHost);
		if (clusterError[0] == errorOld)
			break;
		errorOld = clusterError[0];
		clusterError[0] = 0;
		hipMemcpy(cECuda, clusterError, sizeof(float), hipMemcpyHostToDevice);
		//resetCECKernel<<<1,k>>>(cECCuda, k);
	}
	kmeansGrayOutputKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, outputCuda, cCuda, cCCuda, k);

	/*hipMemcpy(clusterCenters, cCCuda, sizeof(float) * k, hipMemcpyDeviceToHost);
	hipMemcpy(clusterElementCount, cECCuda, sizeof(float) * k, hipMemcpyDeviceToHost);
	for (int i = 0; i < k; i++)
		cout<<clusterCenters[i]<<" "<<clusterElementCount[i] <<endl;*/

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(histCuda);
	hipFree(outputCuda);
	hipFree(cCuda);
	hipFree(cCCuda);
	hipFree(cECCuda);
	hipFree(cCCuda);
	hipFree(cMCuda);
}
__global__ void numberOfColorsKernel(bool* exist, int* number) {
	//if (exist[blockIdx.x] == true) {
	if (exist[threadIdx.x] == true) {
		__syncthreads();
		atomicAdd(number, 1);
		//number[0] += 1;
	}
}

__global__ void compareRGBColorsKernel(float* inputImage, int inputRow, int inputCol, int externalId, bool* exist) {
	int c = blockIdx.x, r = blockIdx.y;
	int id = r * inputCol + c;
	//exist=inputRow*inputCol is all true array for all pixels.if one pixel already exist it is falsed.
	if (id < externalId) {
		if (inputImage[id * 3 + 0] == inputImage[externalId * 3 + 0] && inputImage[id * 3 + 1] == inputImage[externalId * 3 + 1] && inputImage[id * 3 + 2] == inputImage[externalId * 3 + 2])
			exist[externalId] = false;

	}
}

bool* compareRGBColorsCuda(float* inputImage, int inputRow, int inputCol, int* outputHeight) {
	//cout << inputRow * inputCol << endl;//sil
	int externalId;
	bool* exist = (bool*)malloc(sizeof(bool) * inputRow * inputCol);//cuda uzerinde paralel yap
	for (int i = 0; i < inputRow * inputCol; i++)
		exist[i] = true;

	float* inputCuda;
	bool* existCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * 3);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * 3, hipMemcpyHostToDevice);
	hipMalloc(&existCuda, sizeof(bool) * inputRow * inputCol);
	hipMemcpy(existCuda, exist, sizeof(bool) * inputRow * inputCol, hipMemcpyHostToDevice);

	dim3 gridDim;
	for (int i = 0; i < inputRow; i++) {//baslangici kontrol et
		for (int j = 0; j < inputCol; j++) {
			externalId = i * inputCol + j;
			gridDim = dim3(j + 1, i + 1, 1);
			compareRGBColorsKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, externalId, existCuda);
			//cout << "geldi "<< externalId << endl;//sil
		}
	}

	hipMemcpy(exist, existCuda, sizeof(bool) * inputRow * inputCol, hipMemcpyDeviceToHost);

	/*for (int i = 0; i < inputRow; i++) {
		for (int j = 0; j < inputCol; j++) {
			if(exist[i * inputCol + j])
				cout << i <<", "<<j<< "  ";
			//cout << exist[i * inputCol + j] << "  ";
		}
		cout << endl;
	}
	cout << endl;*/

	int* numberCuda, numberOfColors[1] = { 0 };
	hipMalloc(&numberCuda, sizeof(int));
	hipMemcpy(numberCuda, numberOfColors, sizeof(int), hipMemcpyHostToDevice);
	//numberOfColorsKernel << <inputRow * inputCol, 1 >> > (existCuda, numberCuda);
	numberOfColorsKernel << <1, inputRow* inputCol >> > (existCuda, numberCuda);
	hipMemcpy(numberOfColors, numberCuda, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(inputCuda);
	hipFree(existCuda);
	hipFree(numberCuda);

	outputHeight[0] = numberOfColors[0];
	return exist;
	/*float* colors = (float*)malloc(sizeof(float)* numberOfColors[0] *3);
	int id = 0;
	for (int i = 0; i < inputRow * inputCol; i++)
		if (exist[i] == true) {

			colors[id * 3 + 0] = inputImage[i * 3 + 0];
			colors[id * 3 + 1] = inputImage[i * 3 + 1];
			colors[id * 3 + 2] = inputImage[i * 3 + 2];
			id++;
		}

	return colors;*/
}

bool clusterCentersIsSame(float* newC, float* oldC, int colorsHeight) {
	for (int i = 0; i < colorsHeight; i++) {
		float x = newC[i];
		float y = oldC[i];
		//cout << x << " " << y << endl;//sil
		if (newC[i] != oldC[i])
			return false;
	}
	//cout << endl;//sil
	return true;
}

void copyClusterCenters(float* newC, float* oldC, int colorsHeight) {
	for (int i = 0; i < colorsHeight; i++) {
		oldC[i] = newC[i];
	}
}
__global__ void kmeansRGBOutputKernel(float* inputImage, int inputRow, int inputCol, float* outputImage, float* clusters, float* clusterCenters, int k, float* colors, int colorHeight) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		int pixel1 = inputImage[(r * inputCol + c) * 3 + 0];
		int pixel2 = inputImage[(r * inputCol + c) * 3 + 1];
		int pixel3 = inputImage[(r * inputCol + c) * 3 + 2];

		//int cl = clusters[pixel];
		int cl = 0;
		for (int i = 0; i < colorHeight; i++) {
			if (pixel1 == colors[i * 3 + 0] && pixel2 == colors[i * 3 + 1] && pixel3 == colors[i * 3 + 2]) {
				cl = i;
				break;
			}
		}

		outputImage[(r * inputCol + c) * 3 + 0] = clusterCenters[cl * 3 + 0];
		outputImage[(r * inputCol + c) * 3 + 1] = clusterCenters[cl * 3 + 1];
		outputImage[(r * inputCol + c) * 3 + 2] = clusterCenters[cl * 3 + 2];
	}
}

__global__ void kmeansRGBClusterChoiceKernel(float* colors, float* clusters, float* clusterCenters, float* errors, int k) {
	int c = blockIdx.x, r = blockIdx.y;
	//int c = threadIdx.x, r = threadIdx.y;
	//int r = threadIdx.x;
	float rr = colors[r * 3 + 0], gg = colors[r * 3 + 1], bb = colors[r * 3 + 2];
	int minId = 0;
	float min =
		(clusterCenters[0] - rr) * (clusterCenters[0] - rr) +
		(clusterCenters[1] - gg) * (clusterCenters[1] - gg) +
		(clusterCenters[2] - bb) * (clusterCenters[2] - bb);
	float temp;
	for (int i = 1; i < k; i++) {
		temp =
			(clusterCenters[i * 3 + 0] - rr) * (clusterCenters[i * 3 + 0] - rr) +
			(clusterCenters[i * 3 + 1] - gg) * (clusterCenters[i * 3 + 1] - gg) +
			(clusterCenters[i * 3 + 2] - bb) * (clusterCenters[i * 3 + 2] - bb);
		if (temp < min) {
			min = temp;
			minId = i;
		}
	}
	clusters[r] = minId;
	/*__syncthreads();
	atomicAdd(errors, min);*/
	//errors[0] += min;
}

__global__ void kmeansRGBCalcClusterCentersKernel(float* colors, float* clusters, float* clusterCenters, float* clusterMeans, float* clusterElementCount, int k) {
	int c = blockIdx.x, r = blockIdx.y;
	//int c = threadIdx.x, r = threadIdx.y;
	//int r = threadIdx.x;
	int id = clusters[r];

	__syncthreads();
	atomicAdd((clusterMeans + (id * 3 + 0)), colors[r * 3 + 0]);
	atomicAdd((clusterMeans + (id * 3 + 1)), colors[r * 3 + 1]);
	atomicAdd((clusterMeans + (id * 3 + 2)), colors[r * 3 + 2]);
	//clusterMeans[id * 3 + 0] += colors[r * 3 + 0];
	//clusterMeans[id * 3 + 1] += colors[r * 3 + 1];
	//clusterMeans[id * 3 + 2] += colors[r * 3 + 2];

	__syncthreads();
	//clusterElementCount[id] += 1;
	atomicAdd(clusterElementCount + id, 1);

	/*__syncthreads();
	if (r < k) {
		clusterMeans[r * 3 + 0] /= clusterElementCount[r * 3 + 0];
		clusterMeans[r * 3 + 1] /= clusterElementCount[r * 3 + 1];
		clusterMeans[r * 3 + 2] /= clusterElementCount[r * 3 + 2];
		clusterCenters[r * 3 + 0] = clusterMeans[r * 3 + 0];
		clusterCenters[r * 3 + 1] = clusterMeans[r * 3 + 1];
		clusterCenters[r * 3 + 2] = clusterMeans[r * 3 + 2];
		clusterElementCount[r] = 0;
		clusterMeans[r * 3 + 0] = 0;
		clusterMeans[r * 3 + 1] = 0;
		clusterMeans[r * 3 + 2] = 0;
	}*/
}

__global__ void refreshVariables(float* clusterCenters, float* clusterMeans, float* clusterElementCount) {
	int r = threadIdx.x;
	if (clusterElementCount[r] != 0) {
		clusterMeans[r * 3 + 0] /= clusterElementCount[r];
		clusterMeans[r * 3 + 1] /= clusterElementCount[r];
		clusterMeans[r * 3 + 2] /= clusterElementCount[r];
		clusterCenters[r * 3 + 0] = clusterMeans[r * 3 + 0];
		clusterCenters[r * 3 + 1] = clusterMeans[r * 3 + 1];
		clusterCenters[r * 3 + 2] = clusterMeans[r * 3 + 2];
	}
	clusterElementCount[r] = 0;
	clusterMeans[r * 3 + 0] = 0;
	clusterMeans[r * 3 + 1] = 0;
	clusterMeans[r * 3 + 2] = 0;
}

void kmeansRGBCuda(float* inputImage, int inputRow, int inputCol, float* outputImage, int k, float* colors, int colorsHeight) {
	int channel = 3;
	/*for(int i=0;i<5;i++)//sil
		cout << colors[i*3+0]<<" " << colors[i*3+1] << " " << colors[i*3+2] << " " << endl;
	cout << colorsHeight << endl;*/

	float* clusters = (float*)malloc(sizeof(float) * colorsHeight);//holds sets of colors
	float* clusterCenters = (float*)malloc(sizeof(float) * k * channel);
	float* clusterElementCount = (float*)malloc(sizeof(float) * k);
	float* clusterMeans = (float*)malloc(sizeof(float) * k * channel);
	float* clusterError = (float*)malloc(sizeof(float));//sum of all errors
	float errorOld = -1;
	float* clustersOld = (float*)malloc(sizeof(float) * colorsHeight);
	clusterError[0] = 0;
	int mean = 256 / k;
	for (int i = 0; i < k; i++) {
		clusterCenters[i * 3 + 0] = (i)*mean;//rand() % 255;//colors[i * 3 + 0];//
		clusterCenters[i * 3 + 1] = (i)*mean;//rand() % 255;//colors[i * 3 + 1];//
		clusterCenters[i * 3 + 2] = (i)*mean;//rand() % 255;//colors[i * 3 + 2];//
		clusterElementCount[i] = 0;
		clusterMeans[i * 3 + 0] = 0;
		clusterMeans[i * 3 + 1] = 0;
		clusterMeans[i * 3 + 2] = 0;
	}
	for (int i = 0; i < colorsHeight; i++) {
		mean = rand() % k;
		clusters[i] = mean;
		clustersOld[i] = mean;
	}

	/*for (int i = 0; i < k; i++)//sil
		cout << clusterCenters[i * 3 + 0] << " " << clusterCenters[i * 3 + 1] << " " << clusterCenters[i * 3 + 2] << " " << clusterElementCount[i] << endl;
	cout << endl;
	for (int i = 0; i < colorsHeight; i++)//sil
		//cout << clusters[i]<<endl;
		cout << colors[i * 3 + 0] << " " << colors[i * 3 + 1] << " " << colors[i * 3 + 2] << endl;
	cout << endl;*/

	float* inputCuda, * colorsCuda, * outputCuda, * cCuda, * cCCuda, * cECCuda, * cECuda, * cMCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&colorsCuda, sizeof(float) * colorsHeight * channel);
	hipMemcpy(colorsCuda, colors, sizeof(float) * colorsHeight * channel, hipMemcpyHostToDevice);
	hipMalloc(&cCuda, sizeof(float) * colorsHeight);
	hipMemcpy(cCuda, clusters, sizeof(float) * colorsHeight, hipMemcpyHostToDevice);
	hipMalloc(&cCCuda, sizeof(float) * k * channel);
	hipMemcpy(cCCuda, clusterCenters, sizeof(float) * k * channel, hipMemcpyHostToDevice);
	hipMalloc(&cECCuda, sizeof(float) * k);
	hipMemcpy(cECCuda, clusterElementCount, sizeof(float) * k, hipMemcpyHostToDevice);
	hipMalloc(&cMCuda, sizeof(float) * k * channel);
	hipMemcpy(cMCuda, clusterMeans, sizeof(float) * k * channel, hipMemcpyHostToDevice);
	hipMalloc(&cECuda, sizeof(float));
	hipMemcpy(cECuda, clusterError, sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol * channel);

	dim3 gridDim(inputCol, inputRow, 1);
	dim3 gridDim2(1, colorsHeight, 1);
	while (true) {
		kmeansRGBClusterChoiceKernel << <gridDim2, 1 >> > (colorsCuda, cCuda, cCCuda, cECuda, k);
		//kmeansRGBClusterChoiceKernel << <1, gridDim2 >> > (colorsCuda, cCuda, cCCuda, cECuda, k);							
		//kmeansRGBClusterChoiceKernel << <1, colorsHeight >> > (colorsCuda, cCuda, cCCuda, cECuda, k);

		kmeansRGBCalcClusterCentersKernel << <gridDim2, 1 >> > (colorsCuda, cCuda, cCCuda, cMCuda, cECCuda, k);
		//kmeansRGBCalcClusterCentersKernel << <1, gridDim2 >> > (colorsCuda, cCuda, cCCuda, cMCuda, cECCuda, k);	
		//kmeansRGBCalcClusterCentersKernel << <1, colorsHeight >> > (colorsCuda, cCuda, cCCuda, cMCuda, cECCuda, k);

		hipMemcpy(clusters, cCuda, sizeof(float) * colorsHeight, hipMemcpyDeviceToHost);

		/*cout << endl;//sil
		for (int i = 0; i < colorsHeight; i++)
			cout << clusters[i] << endl;
		cout << endl;*/

		if (clusterCentersIsSame(clusters, clustersOld, colorsHeight) == true)
			break;
		copyClusterCenters(clusters, clustersOld, colorsHeight);

		refreshVariables << <1, k >> > (cCCuda, cMCuda, cECCuda);
	}
	kmeansRGBOutputKernel << <gridDim, 1 >> > (inputCuda, inputRow, inputCol, outputCuda, cCuda, cCCuda, k, colorsCuda, colorsHeight);


	/*hipMemcpy(clusters, cCuda, sizeof(float) * colorsHeight, hipMemcpyDeviceToHost);//sil
	for (int i = 0; i < colorsHeight; i++)
		cout << clusters[i] << endl;

	hipMemcpy(clusterCenters, cCCuda, sizeof(float) * k * channel, hipMemcpyDeviceToHost);//sil
	hipMemcpy(clusterElementCount, cECCuda, sizeof(float) * k, hipMemcpyDeviceToHost);
	for (int i = 0; i < k; i++)
		cout << clusterCenters[i*3+0] << " " << clusterCenters[i * 3 + 1] << " " << clusterCenters[i * 3 + 2] << " " << clusterElementCount[i] << endl;
	*/

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol * channel, hipMemcpyDeviceToHost);
	hipFree(colorsCuda);
	hipFree(outputCuda);
	hipFree(cCuda);
	hipFree(cCCuda);
	hipFree(cECCuda);
	hipFree(cCCuda);
	hipFree(cMCuda);
}

__global__ void addImagesGrayKernel(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol, float alpha, float beta) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		outputImage[r * inputCol + c] = inputImage[r * inputCol + c] * alpha + inputImage2[r * inputCol + c] * beta;
	}
}

__global__ void addImagesRGBKernel(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol, float alpha, float beta) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		outputImage[(r * inputCol + c) * 3 + 0] = inputImage[(r * inputCol + c) * 3 + 0] * alpha + inputImage2[(r * inputCol + c) * 3 + 0] * beta;
		outputImage[(r * inputCol + c) * 3 + 1] = inputImage[(r * inputCol + c) * 3 + 1] * alpha + inputImage2[(r * inputCol + c) * 3 + 1] * beta;
		outputImage[(r * inputCol + c) * 3 + 2] = inputImage[(r * inputCol + c) * 3 + 2] * alpha + inputImage2[(r * inputCol + c) * 3 + 2] * beta;
	}
}

void addImagesCuda(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol, float alpha, float beta, int channel) {
	float* inputCuda, * inputCuda2, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&inputCuda2, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda2, inputImage2, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol * channel);

	dim3 gridDim(inputCol, inputRow, 1);
	if (channel == 1) {
		addImagesGrayKernel << <gridDim, 1 >> > (inputCuda, inputCuda2, outputCuda, inputRow, inputCol, alpha, beta);
	}
	else if (channel == 3) {
		addImagesRGBKernel << <gridDim, 1 >> > (inputCuda, inputCuda2, outputCuda, inputRow, inputCol, alpha, beta);
	}

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(inputCuda2);
	hipFree(outputCuda);
}

__global__ void subtractImagesGrayKernel(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		outputImage[r * inputCol + c] = inputImage[r * inputCol + c] - inputImage2[r * inputCol + c];
	}
}

__global__ void subtractImagesRGBKernel(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		outputImage[(r * inputCol + c) * 3 + 0] = inputImage[(r * inputCol + c) * 3 + 0] - inputImage2[(r * inputCol + c) * 3 + 0];
		outputImage[(r * inputCol + c) * 3 + 1] = inputImage[(r * inputCol + c) * 3 + 1] - inputImage2[(r * inputCol + c) * 3 + 1];
		outputImage[(r * inputCol + c) * 3 + 2] = inputImage[(r * inputCol + c) * 3 + 2] - inputImage2[(r * inputCol + c) * 3 + 2];
	}
}

void subtractImagesCuda(float* inputImage, float* inputImage2, float* outputImage, int inputRow, int inputCol, int channel) {
	float* inputCuda, * inputCuda2, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&inputCuda2, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda2, inputImage2, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol * channel);

	dim3 gridDim(inputCol, inputRow, 1);
	if (channel == 1) {
		subtractImagesGrayKernel << <gridDim, 1 >> > (inputCuda, inputCuda2, outputCuda, inputRow, inputCol);
	}
	else if (channel == 3) {
		subtractImagesRGBKernel << <gridDim, 1 >> > (inputCuda, inputCuda2, outputCuda, inputRow, inputCol);
	}

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(inputCuda2);
	hipFree(outputCuda);
}

__global__ void clipGrayImagesKernel(float* inputImage, float* outputImage, int inputRow, int inputCol, int low, int high) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		if (inputImage[r * inputCol + c] > high)
			outputImage[r * inputCol + c] = high;
		else if ((inputImage[r * inputCol + c] < low))
			outputImage[r * inputCol + c] = low;
		else
			outputImage[r * inputCol + c] = inputImage[r * inputCol + c];
	}
}

__global__ void clipRGBImagesKernel(float* inputImage, float* outputImage, int inputRow, int inputCol, int low, int high) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < inputCol && r < inputRow) {
		float rr = inputImage[(r * inputCol + c) * 3 + 0], gg = inputImage[(r * inputCol + c) * 3 + 1], bb = inputImage[(r * inputCol + c) * 3 + 2];

		if (inputImage[(r * inputCol + c) * 3 + 0] > high)
			rr = high;
		else if ((inputImage[(r * inputCol + c) * 3 + 0] < low))
			rr = low;
		if (inputImage[(r * inputCol + c) * 3 + 1] > high)
			gg = high;
		else if ((inputImage[(r * inputCol + c) * 3 + 1] < low))
			gg = low;
		if (inputImage[(r * inputCol + c) * 3 + 2] > high)
			bb = high;
		else if ((inputImage[(r * inputCol + c) * 3 + 2] < low))
			bb = low;

		outputImage[(r * inputCol + c) * 3 + 0] = rr;
		outputImage[(r * inputCol + c) * 3 + 1] = gg;
		outputImage[(r * inputCol + c) * 3 + 2] = bb;
	}
}

void clipImagesCuda(float* inputImage, float* outputImage, int inputRow, int inputCol, int channel, int low, int high) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol * channel);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol * channel, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol * channel);

	dim3 gridDim(inputCol, inputRow, 1);
	if (channel == 1) {
		clipGrayImagesKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputRow, inputCol, low, high);
	}
	else if (channel == 3) {
		clipRGBImagesKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputRow, inputCol, low, high);
	}

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol * channel, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}

__global__ void otsuKernel(float* histogram, int inputRow, int inputCol, float* weightXVariance) {//weightXVariance 254x2
	//2x254
	int c = blockIdx.x, r = blockIdx.y;
	//r cant be 0 or 255

	float weight = 0, mean = 0, variance = 0, colorNum = 0;
	if (c == 0) {
		for (int i = 0; i < r + 1; i++) {
			weight += histogram[i];
			mean += i * histogram[i];
			colorNum += histogram[i];
		}
		if (colorNum == 0)
			colorNum = 1;
		weight = weight / (inputRow * inputCol);
		mean = mean / colorNum;

		for (int i = 0; i < r + 1; i++) {
			variance += (i - mean) * (i - mean) * histogram[i];
		}

	}
	else if (c == 1) {
		for (int i = r + 1; i < 256; i++) {
			weight += histogram[i];
			mean += i * histogram[i];
			colorNum += histogram[i];
		}
		if (colorNum == 0)
			colorNum = 1;
		weight = weight / (inputRow * inputCol);
		mean = mean / colorNum;

		for (int i = r + 1; i < 256; i++) {
			variance += (i - mean) * (i - mean) * histogram[i];
		}
	}
	variance = variance / colorNum;
	weightXVariance[r * 2 + c] = variance * weight;
}

float otsuCuda(float* inputImage, int inputRow, int inputCol) {
	float* histogram = (float*)malloc(sizeof(float) * 256);
	histogramCuda(inputImage, inputRow, inputCol, 1, histogram);
	float* weightXVariance = (float*)malloc(sizeof(float) * 254 * 2);
	float* histCuda, * wXVCuda;

	hipMalloc(&histCuda, sizeof(float) * 256);
	hipMemcpy(histCuda, histogram, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMalloc(&wXVCuda, sizeof(float) * 254 * 2);

	dim3 gridDim(2, 254, 1);
	otsuKernel << <gridDim, 1 >> > (histCuda, inputRow, inputCol, wXVCuda);
	hipMemcpy(weightXVariance, wXVCuda, sizeof(float) * 254 * 2, hipMemcpyDeviceToHost);

	float min = weightXVariance[0 * 2 + 0] + weightXVariance[0 * 2 + 1], minId = 1, temp;
	for (int i = 1; i < 254; i++) {
		temp = weightXVariance[i * 2 + 0] + weightXVariance[i * 2 + 1];
		if (temp < min) {
			min = temp;
			minId = i + 1;
		}
	}

	hipFree(histCuda);
	hipFree(wXVCuda);

	return minId;
}

__global__ void histEqualKernel(float* histogram, int inputRow, int inputCol, float* newColors) {
	int c = blockIdx.x, r = blockIdx.y;
	if (c < 1 && r < 256 && blockIdx.z == 0) {
		float sum = 0;
		for (int i = r; i >= 0; i--) {
			sum += histogram[i];
		}
		sum = sum / (inputRow * inputCol);
		float maxColor = 256.0;
		sum *= (maxColor - 1.0);
		newColors[r] = round(sum);
	}
}

__global__ void histEqualPaintKernel(float* inputImage, int inputRow, int inputCol, float* outputImage, float* newColors) {
	int c = blockIdx.x, r = blockIdx.y;
	int id = inputImage[r * inputCol + c];
	outputImage[r * inputCol + c] = newColors[id];
}

void histEqualCuda(float* inputImage, int inputRow, int inputCol, float* outputImage) {
	float maxColor = 256;
	float* histogram = (float*)malloc(sizeof(float) * maxColor);
	histogramCuda(inputImage, inputRow, inputCol, 1, histogram);
	float* histCuda, * newColorsCuda, * inputCuda, * outputCuda;
	hipMalloc(&histCuda, sizeof(float) * maxColor);
	hipMemcpy(histCuda, histogram, sizeof(float) * maxColor, hipMemcpyHostToDevice);
	hipMalloc(&newColorsCuda, sizeof(float) * maxColor);
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(1, maxColor, 1);
	dim3 gridDim2(inputCol, inputRow, 1);

	histEqualKernel << <gridDim, 1 >> > (histCuda, inputRow, inputCol, newColorsCuda);
	histEqualPaintKernel << <gridDim2, 1 >> > (inputCuda, inputRow, inputCol, outputCuda, newColorsCuda);

	/*float *newColors= (float*)malloc(sizeof(float) * maxColor);
	hipMemcpy(newColors, newColorsCuda, sizeof(float) * maxColor, hipMemcpyDeviceToHost);
	int s = 0;
	for (int i = 0; i < maxColor; i++) {
		cout << i << "  " << histogram[i] << "  " << newColors[i] << endl;
		s += histogram[i];
	}
	cout << inputRow * inputCol<<" "<<s;*/

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
	hipFree(histCuda);
	hipFree(newColorsCuda);
}

__global__ void reverseImageKernel(float* inputImage, float* outputImage, int inputRow, int inputCol) {
	int c = blockIdx.x, r = blockIdx.y;
	outputImage[r * inputCol + c] = 255 - inputImage[r * inputCol + c];
}

void reverseImageCuda(float* inputImage, float* outputImage, int inputRow, int inputCol) {
	float* inputCuda, * outputCuda;
	hipMalloc(&inputCuda, sizeof(float) * inputRow * inputCol);
	hipMemcpy(inputCuda, inputImage, sizeof(float) * inputRow * inputCol, hipMemcpyHostToDevice);
	hipMalloc(&outputCuda, sizeof(float) * inputRow * inputCol);

	dim3 gridDim(inputCol, inputRow, 1);
	reverseImageKernel << <gridDim, 1 >> > (inputCuda, outputCuda, inputRow, inputCol);

	hipMemcpy(outputImage, outputCuda, sizeof(float) * inputRow * inputCol, hipMemcpyDeviceToHost);
	hipFree(inputCuda);
	hipFree(outputCuda);
}